// Heat Transfer Simulation
// MP3, Spring 2016, GPU Programming @ Auburn University
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <omp.h>
#include <hip/hip_runtime.h>

#define N 131072              // Number of points in the rod
#define INITIAL_LEFT  1000.0  // Initial temperature at left end
#define INITIAL_RIGHT 0.0     // Initial temperature at right end
#define ALPHA 0.5             // Constant
#define MAX_TIMESTEPS 10000   // Maximum number of time steps
#define THREADS_PER_BLOCK 256

static void check_result(double *result);

__global__ static void calcHeat(double *ot, double *nt) {
    //for (int i = 1; i < N-1; i++) {
    //    new_t[i] = old_t[i] + ALPHA*(old_t[i-1] + old_t[i+1] - 2*old_t[i]);
    //}
    //new_t[N-1] = old_t[N-1];
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i > 0 && i < N - 1) {
        nt[i] = ot[i] + ALPHA * (ot[i - 1] + ot[i + 1] - 2 * ot[i]);
    }
}
__global__ static void swapTemp(double *ot, double *nt, double *tt) {
    // Swap old and new buffers for next iteration (double-buffering)
    //temp = old_t;
    //old_t = new_t;
    //new_t = temp;
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N) {
        tt[i] = ot[i];
        ot[i] = nt[i];
        nt[i] = ot[i];
    }
}


int main() {
    double *old_t = (double *)malloc(N * sizeof(double));
    double *new_t = (double *)malloc(N * sizeof(double));
    double *temp = (double *)malloc(N * sizeof(double));

    // Initialize arrays/set initial values
    old_t[0] = INITIAL_LEFT;
    for (int i = 1; i < N-1; i++) {
        old_t[i] = 0.0;
    }
    old_t[N-1] = INITIAL_RIGHT;
	new_t[0] = old_t[0];
	new_t[N-1] = old_t[N-1];
    
    int blocksPerGrid = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    
    double start = omp_get_wtime();

    double *d_old, *d_new, *d_temp;
    hipMalloc((void **)&d_old, N * sizeof(double));
    hipMalloc((void **)&d_new, N * sizeof(double));
    hipMalloc((void **)&d_temp, N * sizeof(double));

    hipMemcpy(old_t, d_old, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(new_t, d_new, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(temp, d_temp, N * sizeof(double), hipMemcpyHostToDevice);

    // Compute temperatures at each sample point in the rod over time
    int time;
    for (time = 0; time < MAX_TIMESTEPS; time++) {
        calcHeat<<<blocksPerGrid, THREADS_PER_BLOCK>>>(d_old, d_new);
        swapTemp<<<blocksPerGrid, THREADS_PER_BLOCK>>>(d_old, d_new, d_temp);
    }

    hipMemcpy(old_t, d_old, N * sizeof(double), hipMemcpyDeviceToHost);
    
    hipFree(d_old);
    hipFree(d_new);

    double stop = omp_get_wtime();

    // Show output (final temperatures)
    printf("Stopped after %d time steps\n", time);
    printf("Simulation took %f seconds\n", stop - start);
    
    check_result(old_t);
    return 0;
}

static void check_result(double *result) {
    char output[1024] = { 0 };
    char *out = output;
    
    // Display some of the computed results
    for (int i = 0; i < 6; i++) {
        out += sprintf(out, "%3.3f ", result[i]);
    }
    out += sprintf(out, "... ");
    for (int i = N-6; i < N; i++) {
        out += sprintf(out, "%3.3f ", result[i]);
    }
    printf("Computed: %s\n", output);

    // Display the expected output
    const char *expected = "1000.000 992.021 984.044 976.067 968.095 960.123 ... 0.000 0.000 0.000 0.000 0.000 0.000 ";
    printf("Expected: %s\n", expected);

    // Exit with a nonzero exit code if the two do not match
    assert(strcmp(output, expected) == 0);
}